#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fsyntax-only -verify %s

#include "Inputs/hip/hip_runtime.h"

struct struct_t {};

__global__ void kernel(__grid_constant__ const struct_t arg) {}
__global__ void kernel_non_struct(__grid_constant__ const struct_t arg) {}

__global__ void kernel_non_const(__grid_constant__ struct_t arg) {}

void non_kernel(__grid_constant__ struct_t arg) {}

template <typename T>
__global__ void tk(__grid_constant__ T arg) {}

void foo() {
  tk<const struct_t><<<1,1>>>({});
}
